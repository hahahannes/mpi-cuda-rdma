#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>

int main(int argc, char *argv[])
{
	MPI_Init(&argc, &argv);

	int size;
	MPI_Comm_size(MPI_COMM_WORLD, &size);

	int rank;
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);

	MPI_Status stat;
	int tag1 = 10;
	int tag2 = 20;

	if(size != 2){
		if(rank == 0){
			printf("This program requires exactly 2 MPI ranks, but you are attempting to use %d! Exiting...\n", size);
		}
		MPI_Finalize();
		exit(0);
	}

	size_t N = 6553600;  // 50 MB worth of doubles
	double *DATA = (double*)malloc(N*sizeof(double));
	double *DATA_DEVICE;

	if (DATA == NULL) {
        fprintf(stderr, "Memory allocation failed\n");
        return 1;
    }

    for (size_t i = 0; i < N; i++) {
        DATA[i] = 0.0;
    }

	hipMalloc(&DATA_DEVICE, N*sizeof(double));
	hipMemcpy(DATA_DEVICE, DATA, N*sizeof(double), hipMemcpyHostToDevice);
	
	double start_time, stop_time, elapsed_time;
	start_time = MPI_Wtime();

	if(rank == 0){
		hipMemcpy(DATA, DATA_DEVICE, N*sizeof(double), hipMemcpyDeviceToHost);
		MPI_Send(DATA, N, MPI_DOUBLE, 1, tag1, MPI_COMM_WORLD);
		MPI_Recv(DATA, N, MPI_DOUBLE, 1, tag2, MPI_COMM_WORLD, &stat);
		hipMemcpy(DATA_DEVICE, DATA, N*sizeof(double), hipMemcpyHostToDevice);
	}
	else if(rank == 1){
		MPI_Recv(DATA, N, MPI_DOUBLE, 0, tag1, MPI_COMM_WORLD, &stat);
		hipMemcpy(DATA_DEVICE, DATA, N*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(DATA, DATA_DEVICE, N*sizeof(double), hipMemcpyDeviceToHost);
		MPI_Send(DATA, N, MPI_DOUBLE, 0, tag2, MPI_COMM_WORLD);
	}

	stop_time = MPI_Wtime();
	elapsed_time = stop_time - start_time;
	printf("%.9f\n", elapsed_time);

	hipFree(DATA_DEVICE);
	free(DATA);

	MPI_Finalize();

	return 0;
}
